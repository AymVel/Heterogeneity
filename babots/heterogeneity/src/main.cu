#include "hip/hip_runtime.h"
#include "flamegpu/flamegpu.h"
#include <cmath>

FLAMEGPU_AGENT_FUNCTION(comm_social, flamegpu::MessageSpatial2D, flamegpu::MessageSpatial2D) {
    float radius = 0.5f;

    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");
    const flamegpu::id_t ID = FLAMEGPU->getID();
    // Get previous velocity
    float prev_vx = FLAMEGPU->getVariable<float>("vx");
    float prev_vy = FLAMEGPU->getVariable<float>("vy");

    int count = 0;
    float sumNeighbors = 0; // Initialize sum of neighbors
    for (const auto& message : FLAMEGPU->message_in(x, y)) {
        if (message.getVariable<int>("type") == 1) {
            const float x2 = message.getVariable <float > ("x");
           const float y2 = message.getVariable <float > ("y");

           float x21 = x2 - x;
           float y21 = y2 - y;
           const float separation = sqrt(x21 * x21 + y21 * y21);
            if (separation < radius) {
                const int neighbors = message.getVariable<int>("neighbors");
                sumNeighbors += neighbors; // Add neighbors to sum
                count++; // Increment count of neighbors
            }
        }
    }
    float meanNeighbors = 0.0f;
    if (count > 0) {
        meanNeighbors = sumNeighbors / count; // Calculate mean if count is greater than zero
    }

    // Add persistence factor
    const float persistence = 0.7f;

    // Add some random perturbation to the previous velocity
    float noise_angle = FLAMEGPU->random.uniform<float>() * 3.14159f * 2.0f;
    float noise_vx = cos(noise_angle);
    float noise_vy = sin(noise_angle);

    // Combine persistent velocity and noise
    float vx = persistence * prev_vx + (1 - persistence) * noise_vx;
    float vy = persistence * prev_vy + (1 - persistence) * noise_vy;

    // Normalize the velocity vector
    float length = sqrt(vx * vx + vy * vy);
    vx /= length;
    vy /= length;



     float max = ((radius*2)/0.06f)*((radius*2)/0.06f)*0.75f;
    float step_size =0.15f-meanNeighbors/max;
    if (step_size <0){
        step_size=0;
    }
    x += step_size * vx;
    y += step_size * vy;

    // Ensure the new position is within the environment bounds
    float width = FLAMEGPU->environment.getProperty<float>("ENV_WIDTH");
    // Wrap around if agents move outside the environment bounds
    if (x < 0)
        x += width;
    else if (x >= width)
        x -= width;
    if (y < 0)
        y += width;
    else if (y >= width)
        y -= width;

    // Update agent positions and velocities
    FLAMEGPU->setVariable<float>("x", x);
    FLAMEGPU->setVariable<float>("y", y);
    FLAMEGPU->setVariable<float>("vx", vx);
    FLAMEGPU->setVariable<float>("vy", vy);

    FLAMEGPU -> message_out.setVariable < int > ("id", FLAMEGPU ->getID());
    FLAMEGPU -> message_out.setVariable < int > ("type", FLAMEGPU ->getVariable <int > ("type"));
    FLAMEGPU -> message_out.setLocation(FLAMEGPU -> getVariable <float > ("x"),FLAMEGPU -> getVariable <float > ("y"));

    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(comm_informed, flamegpu::MessageSpatial2D, flamegpu::MessageNone) {
    float radius = 0.5f;

    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");
    const flamegpu::id_t ID = FLAMEGPU->getID();
    // Get previous velocity
    float prev_vx = FLAMEGPU->getVariable<float>("vx");
    float prev_vy = FLAMEGPU->getVariable<float>("vy");

    int count = 0;
    for (const auto & message: FLAMEGPU ->message_in(x, y)) {
      if (message.getVariable <flamegpu::id_t > ("id") != ID) {

       const float x2 = message.getVariable <float > ("x");
       const float y2 = message.getVariable <float > ("y");

       float x21 = x2 - x;
       float y21 = y2 - y;
       const float separation = sqrt(x21 * x21 + y21 * y21);
        if (separation < radius) {
          count++;
        }
      }
    }



    // Add persistence factor
    const float persistence = 0.7f;

    // Add some random perturbation to the previous velocity
    float noise_angle = FLAMEGPU->random.uniform<float>() * 3.14159f * 2.0f;
    float noise_vx = cos(noise_angle);
    float noise_vy = sin(noise_angle);

    // Combine persistent velocity and noise
    float vx = persistence * prev_vx + (1 - persistence) * noise_vx;
    float vy = persistence * prev_vy + (1 - persistence) * noise_vy;

    // Normalize the velocity vector
    float length = sqrt(vx * vx + vy * vy);
    vx /= length;
    vy /= length;



     float max = ((radius*2)/0.06f)*((radius*2)/0.06f)*0.75f;
    float step_size =0.15f-count/max;
    if (step_size <0){
        step_size=0;
    }
    x += step_size * vx;
    y += step_size * vy;

    // Ensure the new position is within the environment bounds
    float width = FLAMEGPU->environment.getProperty<float>("ENV_WIDTH");
    // Wrap around if agents move outside the environment bounds
    if (x < 0)
        x += width;
    else if (x >= width)
        x -= width;
    if (y < 0)
        y += width;
    else if (y >= width)
        y -= width;

    // Update agent positions and velocities
    FLAMEGPU->setVariable<float>("x", x);
    FLAMEGPU->setVariable<float>("y", y);
    FLAMEGPU->setVariable<float>("vx", vx);
    FLAMEGPU->setVariable<float>("vy", vy);
    FLAMEGPU->setVariable<int>("neighbors", count);



    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(output_message,flamegpu::MessageNone,flamegpu::MessageSpatial2D) {
    FLAMEGPU -> message_out.setVariable < int > ("id", FLAMEGPU ->getID());
    FLAMEGPU -> message_out.setVariable < int > ("neighbors", FLAMEGPU ->getVariable <int > ("neighbors"));
FLAMEGPU -> message_out.setVariable < int > ("type", FLAMEGPU ->getVariable <int > ("type"));
    FLAMEGPU -> message_out.setLocation(FLAMEGPU -> getVariable <float > ("x"),FLAMEGPU -> getVariable <float > ("y"));
    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(output_message_social,flamegpu::MessageNone,flamegpu::MessageSpatial2D) {
    FLAMEGPU -> message_out.setVariable < int > ("id", FLAMEGPU ->getID());
    FLAMEGPU -> message_out.setVariable < int > ("neighbors", FLAMEGPU ->getVariable <int > ("neighbors"));
    FLAMEGPU -> message_out.setVariable < int > ("type", FLAMEGPU ->getVariable <int > ("type"));
    FLAMEGPU -> message_out.setLocation(FLAMEGPU -> getVariable <float > ("x"),FLAMEGPU -> getVariable <float > ("y"));
    return flamegpu::ALIVE;
}
FLAMEGPU_INIT_FUNCTION(create_self) {
        const unsigned int AGENT_COUNT = FLAMEGPU -> environment.getProperty < unsigned int > ("AGENT_COUNT");
        const float ENV_WIDTH = FLAMEGPU -> environment.getProperty <float > ("ENV_WIDTH");
        // Create agents
        flamegpu::HostAgentAPI t_pop = FLAMEGPU -> agent("self-informed");
        for (unsigned int i = 0; i < AGENT_COUNT/2; ++i) {
            auto t = t_pop.newAgent();
            t.setVariable<float>("x", FLAMEGPU->random.uniform<float>() * ENV_WIDTH);
            t.setVariable<float>("y", FLAMEGPU->random.uniform<float>() * ENV_WIDTH);
            t.setVariable<float>("vx", 0.0f);
            t.setVariable<float>("vy", 0.0f);
            t.setVariable<int>("neighbors", 0);
            t.setVariable<int>("type", 1);
        }
        flamegpu::HostAgentAPI s_pop = FLAMEGPU -> agent("social");
        for (unsigned int i = 0; i < AGENT_COUNT/2; ++i) {
            auto s = s_pop.newAgent();
            s.setVariable<float>("x", FLAMEGPU->random.uniform<float>() * ENV_WIDTH);
            s.setVariable<float>("y", FLAMEGPU->random.uniform<float>() * ENV_WIDTH);
            s.setVariable<float>("vx", 0.0f);
            s.setVariable<float>("vy", 0.0f);
            s.setVariable<int>("neighbors", 0);
            s.setVariable<int>("type", 0);
        }
}




int main(int argc, const char ** argv) {
    // Define some useful constants
    const float ENV_WIDTH = 20;
    const unsigned int AGENT_COUNT = 1000*ENV_WIDTH;


    // Define the FLAME GPU model
    flamegpu::ModelDescription model("V1.2");


    { // (optional local scope block for cleaner grouping)
        // Define a message of type MessageSpatial2D named location
        flamegpu::MessageSpatial2D::Description message = model.newMessage < flamegpu::MessageSpatial2D > ("information");
        // Configure the message list
        message.setMin(0, 0);
        message.setMax(ENV_WIDTH, ENV_WIDTH);
        message.setRadius(0.5f);
        // Add extra variables to the message
        // X Y (Z) are implicit for spatial messages
        message.newVariable < flamegpu::id_t > ("id");
        message.newVariable <int > ("neighbors");
        message.newVariable <int > ("type");
    }
    { // (optional local scope block for cleaner grouping)
        // Define a message of type MessageSpatial2D named location
        flamegpu::MessageSpatial2D::Description message2 = model.newMessage < flamegpu::MessageSpatial2D > ("location");
        // Configure the message list
        message2.setMin(0, 0);
        message2.setMax(ENV_WIDTH, ENV_WIDTH);
        message2.setRadius(0.5f);
        // Add extra variables to the message
        // X Y (Z) are implicit for spatial messages
        message2.newVariable < flamegpu::id_t > ("id");

        message2.newVariable <int > ("type");
    }

    { // (optional local scope block for cleaner grouping)
        // Define a message of type MessageSpatial2D named location
        flamegpu::MessageSpatial2D::Description message3 = model.newMessage < flamegpu::MessageSpatial2D > ("location2");
        // Configure the message list
        message3.setMin(0, 0);
        message3.setMax(ENV_WIDTH, ENV_WIDTH);
        message3.setRadius(0.5f);
        // Add extra variables to the message
        // X Y (Z) are implicit for spatial messages
        message3.newVariable < flamegpu::id_t > ("id");

        message3.newVariable <int > ("type");
    }
    // Define an agent named self-informed
    flamegpu::AgentDescription self = model.newAgent("self-informed");
    // Assign the agent some variables (ID is implicit to agents, so we don't define it ourselves)
    self.newVariable < float > ("x");
    self.newVariable < float > ("y");
    self.newVariable < float > ("vx");
    self.newVariable < float > ("vy");
    self.newVariable < int > ("neighbors");
    self.newVariable < int > ("type");

    // Define an agent named social
    flamegpu::AgentDescription social = model.newAgent("social");
    // Assign the agent some variables (ID is implicit to agents, so we don't define it ourselves)
    social.newVariable < float > ("x");
    social.newVariable < float > ("y");
    social.newVariable < float > ("vx");
    social.newVariable < float > ("vy");
    social.newVariable < int > ("neighbors");
    social.newVariable < int > ("type");

    // Define agent functions

    flamegpu::AgentFunctionDescription fn_message_social_out = social.newFunction("message_social_out", comm_social);
    fn_message_social_out.setMessageOutput("location");
    fn_message_social_out.setMessageInput("information");


    flamegpu::AgentFunctionDescription fn_message = self.newFunction("message", comm_informed);
    fn_message.setMessageInput("location");
    flamegpu::AgentFunctionDescription fn_message_out = self.newFunction("message_out", output_message);
    fn_message_out.setMessageOutput("information");




    // Define environment properties
    flamegpu::EnvironmentDescription env = model.Environment();
    env.newProperty<unsigned int>("AGENT_COUNT", AGENT_COUNT);
    env.newProperty<float>("ENV_WIDTH", ENV_WIDTH);

    // Define simulation layers


    // Add agent functions to layers
    //fn_message.dependsOn(fn_message_out2);
    fn_message.dependsOn(fn_message_social_out);
    fn_message_social_out.dependsOn(fn_message_out);
    model.addExecutionRoot(fn_message_out);
    //model.addExecutionRoot(fn_message_out2);
    model.generateLayers();




    model.addInitFunction(create_self);






    // Create the simulation
    flamegpu::CUDASimulation cuda_model(model, argc, argv);


    // Only compile this block if being built with visualisation support
#ifdef FLAMEGPU_VISUALISATION
    // Create visualisation
  flamegpu::visualiser::ModelVis m_vis = cuda_model.getVisualisation();


    flamegpu::visualiser::PanelVis ui = m_vis.newUIPanel("Settings");


  // Set the initial camera location and speed
  const float INIT_CAM = ENV_WIDTH / 2.0f;
  m_vis.setInitialCameraTarget(INIT_CAM, INIT_CAM, 0);
  m_vis.setInitialCameraLocation(INIT_CAM, INIT_CAM, ENV_WIDTH);
  m_vis.setCameraSpeed(0.01f);
  m_vis.setSimulationSpeed(24);
  // Add self-informed agents to the visualisation
  flamegpu::visualiser::AgentVis informed_agt = m_vis.addAgent("self-informed");

  // Location variables have names "x" and "y" so will be used by default
  informed_agt.setModel(flamegpu::visualiser::Stock::Models::ICOSPHERE);
  //head is 3 micrometer
  informed_agt.setModelScale(0.03f);

    // Add self-informed agents to the visualisation
  flamegpu::visualiser::AgentVis social_agt = m_vis.addAgent("social");
    social_agt.setColor(flamegpu::visualiser::Stock::Colors::RED);
  // Location variables have names "x" and "y" so will be used by default
 social_agt.setModel(flamegpu::visualiser::Stock::Models::ICOSPHERE);
  //head is 3 micrometer
  social_agt.setModelScale(0.03f);


     // Mark the environment bounds
    flamegpu::visualiser::LineVis pen = m_vis.newPolylineSketch(1, 1, 1, 0.2f);
    pen.addVertex(0, 0, 0);
    pen.addVertex(0, ENV_WIDTH, 0);
    pen.addVertex(ENV_WIDTH, ENV_WIDTH, 0);
    pen.addVertex(ENV_WIDTH, 0, 0);
    pen.addVertex(0, 0, 0);
  // Open the visualiser window
  m_vis.activate();
#endif

    // Run the simulation
    cuda_model.simulate();

#ifdef FLAMEGPU_VISUALISATION
    // Keep the visualisation window active after the simulation has completed
  m_vis.join();
#endif
}